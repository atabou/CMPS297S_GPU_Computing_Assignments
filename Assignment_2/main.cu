
#include "common.h"
#include "timer.h"

void mm_cpu(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K) {
    for (unsigned int row = 0; row < M; ++row) {
        for (unsigned int col = 0; col < N; ++col) {
            float sum = 0.0f;
            for(unsigned int i = 0; i < K; ++i) {
                sum += A[row*K + i]*B[i*N + col];
            }
            C[row*N + col] = sum;
        }
    }
}

int main(int argc, char**argv) {

    hipDeviceSynchronize();

    // Allocate memory and initialize data
    Timer timer;
    unsigned int M = (argc > 1)?(atoi(argv[1])):200;
    unsigned int N = (argc > 2)?(atoi(argv[2])):500;
    unsigned int K = (argc > 3)?(atoi(argv[3])):1000;
    float* A = (float*) malloc(M*K*sizeof(float));
    float* B = (float*) malloc(K*N*sizeof(float));
    float* C_cpu = (float*) malloc(M*N*sizeof(float));
    float* C_gpu = (float*) malloc(M*N*sizeof(float));
    for (unsigned int row = 0; row < M; ++row) {
        for (unsigned int col = 0; col < K; ++col) {
            A[row*K + col] = 1.0*rand()/RAND_MAX;
        }
    }
    for (unsigned int row = 0; row < K; ++row) {
        for (unsigned int col = 0; col < N; ++col) {
            B[row*N + col] = 1.0*rand()/RAND_MAX;
        }
    }

    // Compute on CPU
    startTime(&timer);
    mm_cpu(A, B, C_cpu, M, N, K);
    stopTime(&timer);
    printElapsedTime(timer, "CPU time", CYAN);

    // Compute on GPU
    startTime(&timer);
    mm_gpu(A, B, C_gpu, M, N, K);
    stopTime(&timer);
    printElapsedTime(timer, "GPU time", DGREEN);

    // Verify result
    for (unsigned int row = 0; row < M; ++row) {
        for (unsigned int col = 0; col < N; ++col) {
            float diff = (C_cpu[row*N + col] - C_gpu[row*N + col])/C_cpu[row*N + col];
            const float tolerance = 0.00001;
            if(diff > tolerance || diff < -tolerance) {
                printf("Mismatch at row %u, col %u (CPU result = %e, GPU result = %e)\n", row, col, C_cpu[row*N + col], C_gpu[row*N + col]);
                exit(0);
            }
        }
    }

    // Free memory
    free(A);
    free(B);
    free(C_cpu);
    free(C_gpu);

    return 0;

}

