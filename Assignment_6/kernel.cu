#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

#define BLOCK_DIM 1024

__global__ void scan_kernel(float* input, float* output, float* partialSums, unsigned int N) {

    // TODO




















}

__global__ void add_kernel(float* output, float* partialSums, unsigned int N) {

    // TODO







}

void scan_gpu_d(float* input_d, float* output_d, unsigned int N) {

    Timer timer;

    // Configurations
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    const unsigned int numElementsPerBlock = 2*numThreadsPerBlock;
    const unsigned int numBlocks = (N + numElementsPerBlock - 1)/numElementsPerBlock;

    // Allocate partial sums
    startTime(&timer);
    float *partialSums_d;
    hipMalloc((void**) &partialSums_d, numBlocks*sizeof(float));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Partial sums allocation time");

    // Call kernel
    startTime(&timer);
    scan_kernel <<< numBlocks, numThreadsPerBlock >>> (input_d, output_d, partialSums_d, N);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Scan partial sums then add
    if(numBlocks > 1) {

        // Scan partial sums
        scan_gpu_d(partialSums_d, partialSums_d, numBlocks);

        // Add scanned sums
        add_kernel <<< numBlocks, numThreadsPerBlock >>> (output_d, partialSums_d, N);

    }

    // Free memory
    startTime(&timer);
    hipFree(partialSums_d);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

